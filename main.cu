#include "print.h"
#include "stopwatch.h"
#include "transmit.h"
#include "brainfuck.h"
#include <unistd.h>

static const int F_CPU = 0x01;
static const int F_TIME = 0x02;
static const int F_MEMCPY_TIME = 0x04;
static int flag = 0;

__host__ void kernel_brainfuck(char **res, char *source, int source_len){
    char *source_d, *res_d;

    if(flag & F_TIME && flag & F_MEMCPY_TIME){
        stop_watch_start();
    }
    transmit_data(&source_d, source, source_len);

    hipMalloc(&res_d, sizeof(char) * *source);
    if(flag & F_TIME && !(flag & F_MEMCPY_TIME)){
        stop_watch_start();
    }
    kernel<<<1, *source>>>(res_d, source_d);
    if(flag & F_TIME && !(flag & F_MEMCPY_TIME)){
        stop_watch_stop();
    }
    hipFree(source_d);

    hipMemcpy(*res, res_d, sizeof(char) * *source, hipMemcpyDeviceToHost);
    hipFree(res_d);
    if(flag & F_TIME && flag & F_MEMCPY_TIME){
        stop_watch_stop();
    }
}

__host__ void host_brainfuck(char **res, char *source){
    if(flag & F_TIME){
        stop_watch_start();
    }
    host(*res, source);
    if(flag & F_TIME){
        stop_watch_stop();
    }
}

__host__ int main(int argc, char *argv[]){
    extern int optind, optopt;
    extern int opterr;
    FILE *in;
    char c;
    Source *source;
    char *packed_source;
    int packed_source_len;

    opterr = 0;
    while((c = getopt(argc, argv, "chmtv")) != -1){
        switch(c){
            case 'c':
                flag = flag | F_CPU;
                break;
            case 'h':
                help();
            case 'm':
                flag = flag | F_MEMCPY_TIME;
            case 't':
                flag = flag | F_TIME;
                break;
            case 'v':
                version();
            default:
                error("illigal option \"%c\".\n%s\n", optopt, usage);
        }
    }
    argc -= optind;
    argv += optind;

    in = (argc > 0) ? fopen(argv[0], "r") : stdin;
    if(in == NULL){
        error("There is no file named \"%s\".\n", argv[0]);
    }

    source = get_strings(in);
    fclose(in);
    // TEST >>>>>
    /*
    Source *source_tmp;
    Code *code;
    puts("\n");
    source_tmp = source;
    while(source){
        code = source->codes;
        while(code){
            puts(code->code);
            code = code->next;
        }
        printf("%d\n", source->codes_len);
        source = source->next;
    }
    source = source_tmp;
    */
    // <<<<< TEST
    packed_source_len = pack_strings(&packed_source, source);
    // TEST >>>>>
    /*
    printf("\n\n%d\n\n", packed_source_len);
    int i;
    for(i = 0; i < packed_source_len; i++){
        if(packed_source[i] < 33){
            printf("%d ", packed_source[i]);
        } else{
            printf("%c", packed_source[i]);
        }
    }
    puts("");
    */
    // <<<<< TEST
    char *res = (char *)malloc(sizeof(char) * *packed_source);
    if(flag & F_CPU){
        host_brainfuck(&res, packed_source);
    } else{
        kernel_brainfuck(&res, packed_source, packed_source_len);
    }
    puts(res);
    // TEST >>>>>
    /*
    for(i = 0; i < *packed_source; i++){
        if(res[i] < 33){
            printf("%d ", res[i]);
        } else{
            printf("%c", res[i]);
        }
    }
    */
    // <<<<< TEST
    if(flag & F_TIME){
        printf("\nReal run time: %10.6f (sec)\n", get_stop_watch_time());
    }

    return EXIT_SUCCESS;
}
