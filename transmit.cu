#include "transmit.h"

/*
 * [WARNING] String length must be shorter than 255.
 *
 * @param::dist   data : *(char<fold(+, map(length, strs)) + len>)
 * @param::source strs : { char[], char[], ... }
 * @param::source len  : length(strs)
 */
__host__ int pack_strings(char *data[], char *strs[], char len){
    char i, j;
    char *strhead, *lenhead;
    int data_len;

    **data = len;
    lenhead = *data + 1;
    strhead = lenhead + len;
    data_len = len + 1;

    for(i = 0; i < len; i++){
        for(j = 0; strs[i][j]; j++){
            *strhead++ = strs[i][j];
        }
        *strhead++ = '\0';
        *lenhead++ = j + 1;
        data_len += j + 1;
    }

    return data_len;
}

__host__ void transmit_data(char **data_d, char *data, int len){
    hipMalloc(data_d, sizeof(char) * len);
    hipMemcpy(*data_d, data, sizeof(char) * len, hipMemcpyHostToDevice);
}
