#include "transmit.h"

__host__ Code *create_code(){
    Code *code = (Code *)malloc(sizeof(Code));
    code->next = NULL;
    return code;
}

__host__ Source *create_source(){
    Source *source = (Source *)malloc(sizeof(Source));
    source->codes = create_code();
    source->codes_len = 0;
    source->next = NULL;
    return source;
}

__host__ void deleteSource(Source *source){
    Source *next;
    Code *next_code;

    while(source){
        next = source->next;
        while(source->codes){
            next_code = source->codes->next;
            free(source->codes);
            source->codes = next_code;
        }
        free(source);
        source = next;
    }
}

__host__ Source *get_strings(FILE *in){
    Source *source = create_source();
    Source *cur_source = source;
    Source *prev_source = NULL;
    Code *code = cur_source->codes;
    int i = 0;
    char c;

    for(c = fgetc(in); c != EOF; c = fgetc(in)){
        if(c == '\n'){
            if(cur_source->codes_len > 0){
                code->code[i] = '\0';
                cur_source->next = create_source();
                prev_source = cur_source;
                cur_source = cur_source->next;
                code = cur_source->codes;
                i = 0;
            }
        } else{
            if(i == CODE_LENGTH){
                puts("overflow. malloc next.");
                code->next = create_code();
                code = code->next;
                i = 0;
            }
            code->code[i++] = c;
            cur_source->codes_len++;
        }
    }
    if(cur_source->codes_len == 0 && prev_source != NULL){
        prev_source->next = NULL;
        deleteSource(cur_source);
    }
    return source;
}

/*
 * [WARNING] String length must be shorter than 255.
 */
__host__ int pack_strings(char **data, Source *source){
    Source *cur_source;
    int source_len = 0, data_len = 0;
    char i;
    char *strhead, *lenhead;

    cur_source = source;
    while(cur_source){
        source_len++;
        data_len += cur_source->codes_len;
        cur_source = cur_source->next;
    }
    data_len += source_len * 2 + 1;

    *data = (char *)malloc(sizeof(char) * data_len);
    **data = source_len;
    lenhead = *data + 1;
    strhead = lenhead + source_len;

    while(source){
        *lenhead++ = source->codes_len + 1;
        while(source->codes){
            for(i = 0; i < CODE_LENGTH && source->codes->code[i]; i++){
                *strhead++ = source->codes->code[i];
            }
            *strhead++ = '\0';
            source->codes = source->codes->next;
        }
        source = source->next;
    }

    return data_len;
}

__host__ void transmit_data(char **data_d, char *data, int len){
    hipMalloc(data_d, sizeof(char) * len);
    hipMemcpy(*data_d, data, sizeof(char) * len, hipMemcpyHostToDevice);
}
