#include "hip/hip_runtime.h"
#include "brainfuck.h"

__host__ void host(char *res, char *data){
    int thread_size = *data;
    int phead;
    int i, idx;

    for(idx = 0; idx < thread_size; idx++){
        phead = *data + 1;
        for(i = 0; i < idx; i++){
            phead += data[i + 1];
        }
        res[idx] = brainfuck(data + phead, data[idx + 1]);
    }
}

__global__ void kernel(char *res, char *data){
    int idx = threadIdx.x;
    int phead = *data + 1;
    int i;

    for(i = 0; i < idx; i++){
        phead += data[i + 1];
    }
    res[idx] = brainfuck(data + phead, data[idx + 1]);
}

__host__ __device__ char brainfuck(char *source, int len){
    Expression *ex = parse(&source, EOP);
    if(ex == NULL){
        return '\0';
    }
    return run(ex);
}

__host__ __device__ void appendExpression(Expression *head, Expression *append){
    append->prev = head->prev;
    append->prev->next = append;
    head->prev = append;
}

/*
 * compile
 */
__host__ __device__ Expression *createAtomExpression(ExpressionKind kind, int value){
    Expression *ex = (Expression *)malloc(sizeof(Expression));
    ex->kind = kind;
    ex->u.value = value;
    ex->next = NULL;
    ex->prev = NULL;
    return ex;
}
__host__ __device__ void addAtomExpression(Expression **head, ExpressionKind kind, int value){
    if(*head == NULL){
        *head = createAtomExpression(kind, value);
        (*head)->prev = *head;
        return;
    }
    if((*head)->prev->kind == kind){
        (*head)->prev->u.value += value;
    } else{
        appendExpression(*head, createAtomExpression(kind, value));
    }
}

__host__ __device__ Expression *createWhileExpression(){
    Expression *ex = (Expression *)malloc(sizeof(Expression));
    ex->kind = WHILE_EXPRESSION;
    ex->u.block = NULL;
    ex->next = NULL;
    ex->prev = NULL;
    return ex;
}
__host__ __device__ void addWhileExpression(Expression **head){
    if(*head == NULL){
        *head = createWhileExpression();
        (*head)->prev = *head;
        return;
    }
    appendExpression(*head, createWhileExpression());
}

__host__ __device__ Token lex(char **source){
    switch(*(*source)++){
        case '+':
            return INC;
        case '-':
            return DEC;
        case '>':
            return NEXT;
        case '<':
            return PREV;
        case '.':
            return PUT;
        case ',':
            return GET;
        case '[':
            return BEGIN;
        case ']':
            return END;
        case '\0':
            return EOP;
        default:
            return lex(source);
    }
}

__host__ __device__ Expression *parse(char **source, Token period){
    Token token;
    Expression *head = NULL;

    while((token = lex(source)) != period){
        switch(token){
            case INC:
                addAtomExpression(&head, ADD_EXPRESSION, 1);
                break;
            case DEC:
                addAtomExpression(&head, ADD_EXPRESSION, -1);
                break;
            case NEXT:
                addAtomExpression(&head, MOVE_EXPRESSION, 1);
                break;
            case PREV:
                addAtomExpression(&head, MOVE_EXPRESSION, -1);
                break;
            case GET:
                addAtomExpression(&head, GET_EXPRESSION, 1);
                break;
            case PUT:
                addAtomExpression(&head, PUT_EXPRESSION, 1);
                break;
            case BEGIN:
                addWhileExpression(&head);
                head->prev->u.block = parse(source, END);
                break;
            case END:
            case EOP:
                return NULL;
        }
    }
    return head;
}

/*
 * run
 */

#define MEM_GRID_X 10
#define DEFAULT_MEM_SIZE 20
__host__ __device__ void reallocVMMemory(VirtualMachine *vm, int size){
    int i, j;
    int required_y = (size - 1) / MEM_GRID_X + 1;
    int **new_memory = (int **)malloc(sizeof(int *) * required_y);

    for(i = 0; i < required_y; i++){
        if(vm->memory != NULL && vm->memory_size >= (i + 1) * MEM_GRID_X){
            new_memory[i] = vm->memory[i];
        } else{
            new_memory[i] = (int *)malloc(sizeof(int) * MEM_GRID_X);
            for(j = 0; j < MEM_GRID_X; j++){
                new_memory[i][j] = 0;
            }
        }
    }
    vm->memory = new_memory;
    vm->memory_size = size;
}

__host__ __device__ VirtualMachine *createVM(Expression *program){
    VirtualMachine *vm = (VirtualMachine *)malloc(sizeof(VirtualMachine));

    vm->program = program;
    vm->header = 0;
    vm->memory = NULL;
    reallocVMMemory(vm, DEFAULT_MEM_SIZE);

    return vm;
}

__host__ __device__ void deleteExpression(Expression *ex){
    while(ex->next){
        ex = ex->next;
        free(ex->prev);
    }
    free(ex);
}

__host__ __device__ void deleteVM(VirtualMachine *vm){
    int i;
    int y = vm->memory_size / MEM_GRID_X;
    for(i = 0; i < y; i++){
        free(vm->memory[i]);
    }
    free(vm->memory);
    free(vm);
}

__host__ __device__ int *seekCurrentVMMemory(VirtualMachine *vm){
    int y = vm->header / MEM_GRID_X;
    int x = vm->header - MEM_GRID_X * y;
    return vm->memory[y] + x;
}

__host__ __device__ void addVMMemory(VirtualMachine *vm, int increment){
    *seekCurrentVMMemory(vm) += increment;
}

__host__ __device__ void moveVMHeader(VirtualMachine *vm, int increment){
    vm->header += increment;
    if(vm->header >= vm->memory_size){
        reallocVMMemory(vm, vm->header + 1);
    }
}

__host__ __device__ int getVMValue(VirtualMachine *vm){
    return *seekCurrentVMMemory(vm);
}

__host__ __device__ int runVM(VirtualMachine *vm, int ret){
    Expression *jumped;

    while(vm->program != NULL){
        switch(vm->program->kind){
            case ADD_EXPRESSION:
                addVMMemory(vm, vm->program->u.value);
                break;
            case MOVE_EXPRESSION:
                moveVMHeader(vm, vm->program->u.value);
                break;
            case GET_EXPRESSION:
                // TODO
                break;
            case PUT_EXPRESSION:
                ret = getVMValue(vm);
                break;
            case WHILE_EXPRESSION:
                if(getVMValue(vm) != 0){
                    jumped = vm->program;
                    vm->program = vm->program->u.block;
                    ret = runVM(vm, ret);
                    vm->program = jumped;
                    continue;
                }
        }
        vm->program = vm->program->next;
    }

    return ret;
}

__host__ __device__ int run(Expression *program){
    VirtualMachine *vm = createVM(program);
    int ret = runVM(vm, 0);
    deleteExpression(program);
    deleteVM(vm);
    return ret;
}
